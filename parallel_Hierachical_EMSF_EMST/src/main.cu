#include <iostream>
#include "ConfigParams.h"
#include "Node.h"
#include "GridOfNodes.h"
#include "NIter.h"
#include "ViewGrid.h"


#include "TestMstBoruvka.h"

using namespace std;
using namespace components;
using namespace meshing;

#define TEST_CODE  0
#define SECTION_PARAMETRES  0

int main(int argc, char *argv[])
{
    char* fileData;
    char* fileSolution;
    char* fileStats;
    char* fileConfig;
    //! qiao 092015 add
    char* fileCalibSt;

    /*
     * Lecture des fichiers d'entree
     */
    if (argc <= 1)
    {
        fileData = "input.data";
        fileSolution = "output.data";
        fileStats = "output.stats";
        fileConfig = "config.cfg";
        fileCalibSt = "calib.txt";
    }
    else
        if (argc == 2)
        {
            fileData = argv[1];
            fileSolution = "output.data";
            fileStats = "output.stats";
            fileConfig = "config.cfg";
            fileCalibSt = "calib.txt";
        }
        else
            if (argc == 3)
            {
                fileData = argv[1];
                fileSolution = argv[2];
                fileStats = "output.stats";
                fileConfig = "config.cfg";
                fileCalibSt = "calib.txt";
            }
            else
                if (argc == 4)
                {
                    fileData = argv[1];
                    fileSolution = argv[2];
                    fileStats = argv[3];
                    fileConfig = "config.cfg";
                    fileCalibSt = "calib.txt";
                }
                else
                    if (argc >= 5)
                    {
                        fileData = argv[1];
                        fileSolution = argv[2];
                        fileStats = argv[3];
                        fileConfig = argv[4];
                        fileCalibSt = argv[5];
                    }

    /*
     * Lecture des parametres
     */
    ConfigParams params(fileConfig);
    params.readConfigParameters();



    if (params.functionModeChoice == 1) {
        cout << "TEST Parallel Euclidean Minimum Spanning Forest: " << endl;
        TestMstBoruvka t(fileData, fileSolution, fileStats, params);
        t.initialize();
        t.run();
        cout << "Fin de test " << params.functionModeChoice << endl;
    }

    return 0;
}//main

